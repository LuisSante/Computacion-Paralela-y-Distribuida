#include "hip/hip_runtime.h"
﻿#include "lodepng.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include <iostream>
#define BLUR_SIZE 7
#define R 0
#define G 1
#define B 2
#define A 3

__global__ void greyKernel(unsigned char* in, unsigned char* out, int width, int height, int num_channel, int channel) {

  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if(col < width && row < height) {
    int greyOffset = row*width + col; //indice de pixel para el out
    int rgbOffset = greyOffset*num_channel;
    unsigned char r = in[rgbOffset ]; // red value for pixel
    unsigned char g = in[rgbOffset + 2]; // green value for pixel
    unsigned char b = in[rgbOffset + 3]; // blue value for pixel

    out[row * width * num_channel + col * num_channel + channel] = 0.21f*r + 0.71f*g + 0.07f*b;
  }
}

int main() {

  int width, height,n;
  unsigned char *image = stbi_load("lena_original.jpg",&width,&height,&n,0);
  unsigned char *output = (unsigned char*)malloc(width * height * n *sizeof(unsigned char));
  unsigned char* Dev_Input_Image = NULL;
  unsigned char* Dev_Output_Image = NULL;
  hipMalloc((void**)&Dev_Input_Image, sizeof(unsigned char)* height * width * n);
  hipMalloc((void**)&Dev_Output_Image, sizeof(unsigned char)* height * width * n);

  hipMemcpy(Dev_Input_Image, image, sizeof(unsigned char) * height * width * n, hipMemcpyHostToDevice);

  //kernel call
  dim3 blockSize(16, 16, 1);
  dim3 gridSize(width/blockSize.x, height/blockSize.y,1);
  greyKernel <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height,n,R);
  greyKernel <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height,n,G);
  greyKernel <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height,n,B);
  
  hipDeviceSynchronize();

    hipMemcpy(image, Dev_Output_Image, sizeof(unsigned char) * height * width * n, hipMemcpyDeviceToHost);
  hipFree(Dev_Input_Image);
  hipFree(Dev_Output_Image);
  stbi_write_png("lena_original_grey.jpg", width, height, n, image, width * n);


  return 0;
}